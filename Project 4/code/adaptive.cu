/*
 * Adaptive Non Local Means - Parallel Implementation
 *
 * Authors: Athanasiadis Christos athanasc@ece.auth.gr AEM 8416
 *     	    Matsoukas Vasileios   vmatsouk@ece.auth.gr AEM 8743
 *
 *  adaptive NLM using loop for each region
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/wait.h>
#include <sys/time.h>
#include <math.h>
#include <string.h>

#define MAX 512

struct timeval startwtime, endwtime;
float seq_time=0;
float* image_pad(float *im, int m, int n, int patchsize_x, int patchsize_y);
float* gauss_patch(float* gaussian, int offsetx, int offsety, float patchSigma);


//------------Kernel functions------------//

__global__ void neighbs(float* im, float* neighbors, float* gauss, int offsety, int offsetx, int patch_size, int newpatch_size, int s_x, int s_y, int patchsize_y){
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	int m = s_y - 2*offsety;
	int n = s_x - 2*offsetx;
	if (id<m*n){
		int kx = blockIdx.y-offsetx;
		int ky = blockIdx.z-offsety;
		int neighbor_id = blockIdx.y*patchsize_y+blockIdx.z;
		neighbors[id*newpatch_size+neighbor_id] = im[(offsety+ky+id%m)*s_x+kx+offsetx+(id/m)]*gauss[neighbor_id];

	}
}


__global__ void affinity(float *image, float *neighbors, int pixels, int patchsize, float std, int offsetx,int offsety,int patchsize_y, float *row, int* reg){
	__shared__ float Ys[16][16];
	__shared__ float Xs[16][16];
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	int yBegin = by * 16 * patchsize;
	int xBegin = bx * 16 * patchsize;
	int yEnd = yBegin + patchsize - 1, y, x, k;
	float tmp, c = 0 , s = 0;

	int pixel_x = bx*16 + tx;
	int pixel_y = by*16 + ty;

	int t = 0;
	for(y=yBegin,x=xBegin; y<=yEnd; y+=16,x+=16){

		if (pixel_y < pixels) Ys[ty][tx] = neighbors[reg[pixel_y]*patchsize + tx + t*16];
		else Ys[ty][tx] = 0;
		if (bx*16+ty < pixels) Xs[tx][ty] = neighbors[reg[bx*16 + ty]*patchsize + tx + t*16];
		else Xs[tx][ty] = 0;

		t++;

		__syncthreads();

		for(k=0;k<16;k++){
			tmp = Ys[ty][k] - Xs[k][tx];
			s += tmp*tmp;
		}
		__syncthreads();
	}



	if (pixel_y < pixels && pixel_x < pixels){
		Xs[ty][tx] = exp(-s/std);
		Ys[ty][tx] = Xs[ty][tx]*neighbors[reg[pixel_x]*patchsize+patchsize_y*offsetx+offsety];
	}
	else {
		Ys[ty][tx] = 0;
		Xs[ty][tx] = 0;
	}
	__syncthreads();

	if (pixel_y< pixels && tx==0){
		s=0;
		for (k=0; k<16; k++){
			c+=Ys[ty][k];
			s+=Xs[ty][k];
		}
		atomicAdd(&image[reg[pixel_y]], c); atomicAdd(&row[reg[pixel_y]],s);
	}

}


__global__ void newimage(float* im, float* row, int k, int n){
	int id = blockIdx.x*n + blockIdx.y;
	if (id < k) im[id] = im[id]/row[id];
}



int main(int argc, char **argv){

	if (argc!=5){
		printf("Wrong number of inputs.\n");
		printf("Usage: %s binary_input binary_output image_rows image_columns \n",argv[0]);
		exit(0);
	}


	FILE *fp;  //file pointer to the binary image file
	int m = atoi(argv[3]); //image rows
	int n = atoi(argv[4]); //image columns

	int patchsize_x = 5, patchsize_y = 5;
	int patch_size = patchsize_x*patchsize_y;
	float patchSigma = 1.6667;
	int nLevel = 6;


	int size_y = m + (patchsize_y-1);
	int offsety = (patchsize_y-1)/2;
	int size_x = n + (patchsize_x-1);
	int offsetx = (patchsize_x-1)/2;

	float *im,*new_im, *gaussian;
	float *d_im,*d_new_im, *d_gaussian, *d_neighbors, *d_row;
	int blocksx,blocksy,blocksz;
	int* L = ( int *)malloc(m*n*sizeof( int *));


	float* mean = (float *)calloc(nLevel,sizeof( float ));
	float* std = (float *)calloc(nLevel,sizeof( float ));
	int* count = (int *)calloc(nLevel,sizeof(int));

	//Allocate (1-D) memory for input image and filtered image
	im = ( float *)malloc(size_y*size_x*sizeof( float ));
	new_im=(float *)malloc(m*n*sizeof(float));

	//Open data file
	fp=fopen(argv[1],"rb");

	if (!fp){
		printf("Unable to open file!");
		return 1;
	}

	int index=0;

	//Store input image data and find sigma for each region
	for (int i=offsety; i<size_y-offsety; i++){
    		fread(im+i*size_x+offsetx,sizeof(float),n,fp);
		for (int j=0; j<n; j++){
			index= j*m+(i-offsety);
			L[index] = round((nLevel-1)*im[(index%m+offsety)*size_x + index/m +offsetx]);
			mean[L[index]]+= im[(index%m+offsety)*size_x + index/m +offsetx];
			count[L[index]]++;
		}
	}

	fclose(fp);

	int** regions = ( int **)malloc(nLevel*sizeof( int *));
	for (int i=0; i<nLevel; i++)
		regions[i] = ( int *)malloc(count[i]*sizeof(int));
	int* regionscount = (int *)calloc(nLevel,sizeof(int));
	for (int i=0; i<nLevel; i++) mean[i]/=count[i];
	int reg;
	for (int j=0; j<n; j++){
		for (int i=0; i<m; i++){
			index= j*m+i;
			reg = L[index];
			std[reg]+=(im[(index%m+offsety)*size_x + index/m +offsetx]-mean[reg])*(im[(index%m+offsety)*size_x + index/m +offsetx]-mean[reg]);
			regions[reg][regionscount[reg]++] = index;

		}
	}

	for (int i=0; i<nLevel; i++) std[i]/=(count[i]-1);

	//Pad image's borders symmetrically
	im=image_pad(im,m,n,patchsize_x,patchsize_y);

	//Create gaussian patch
	gaussian = ( float *)malloc(patch_size*sizeof( float));
	gaussian = gauss_patch(gaussian,offsetx,offsety,patchSigma);

	/////// Set new patchsize /////////
	int newpatch_size = patch_size;
	if (patch_size%16!=0) newpatch_size = (patch_size/16 + 1)*16;


	//Allocate space for device copies
	hipMalloc((void **)&d_im, size_x*size_y*sizeof(float));
	hipMalloc((void **)&d_new_im, m*n*sizeof(float));
	hipMalloc((void **)&d_row, m*n*sizeof(float));
	hipMalloc((void **)&d_neighbors, (m*n)*newpatch_size*sizeof(float));
	hipMalloc((void **)&d_gaussian, patch_size*sizeof(float));


	//Copy inputs to device
	hipMemcpy(d_im, im, size_x*size_y*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gaussian, gaussian, patch_size*sizeof(float), hipMemcpyHostToDevice);



	free(mean);free(gaussian);free(im);free(L);

	/////// Cuda Memset /////////
	hipMemset(d_neighbors, 0, (m*n)*newpatch_size*sizeof(float));
	hipMemset(d_row, 0, (m*n)*sizeof(float));
	hipMemset(d_new_im, 0, (m*n)*sizeof(float));

	//Finding neighbors, number_of_neighbors % 16 = 0

	blocksx = (m*n)/MAX;
	if ((m*n)%MAX!=0)
		blocksx++;

	blocksy = patchsize_x;
	blocksz = patchsize_y;

	//Begin Chronometer
	gettimeofday (&startwtime, NULL);

	neighbs<<<dim3(blocksx,blocksy,blocksz),MAX>>>(d_im, d_neighbors, d_gaussian, offsety, offsetx, patch_size, newpatch_size, size_x, size_y, patchsize_y);


	int k;

	for (int q=0; q<nLevel; q++){
		int *d_region;
		k = count[q];
		hipMalloc((void **)&d_region, k*sizeof(int));
		hipMemcpy(d_region, regions[q] , k*sizeof(int), hipMemcpyHostToDevice);



		//Finding the affinity matrix, k%16 = 0

		blocksx = k/16;

		if (k%16!=0) blocksx = (blocksx/16+1)*16;
		blocksy = blocksx;

		affinity<<<dim3(blocksx, blocksy, 1),dim3(16, 16, 1)>>>(d_new_im, d_neighbors, k, newpatch_size, std[q], offsetx, offsety, patchsize_y, d_row, d_region);

		hipFree(d_region);

	}

	newimage<<<dim3(m, n), 1>>>(d_new_im, d_row, m*n, n);

	//Stop chronometer
	hipDeviceSynchronize();
	gettimeofday (&endwtime, NULL);

	printf("\n");
	seq_time = (float)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6+ endwtime.tv_sec - startwtime.tv_sec);
	printf("Total time needed using adaptive nlmeans: %f secs\n", seq_time);


    	hipMemcpy(new_im,d_new_im, m*n*sizeof(float), hipMemcpyDeviceToHost );

//    	printf("Thats the filtered image:\n");
//
//
//		for (int j=0; j<m; j++){
//			for (int i=0; i<n ; i++)
//	       			printf( "%f ",new_im[i*m+j]);
//
//	  	        printf("\n\n");
//	    }

		//Save output to a binary file
		fp=fopen(argv[2],"w");
		for (int j=0; j<m; j++){
			for (int i=0; i<n ; i++)
				fwrite(&new_im[i*m+j],sizeof(float),1,fp);
	    }
		fclose(fp);



	//Free allocated memory
	hipFree(d_im); hipFree(d_gaussian); hipFree(d_neighbors); hipFree(d_new_im); hipFree(d_row);

	free(std); free(new_im);

	return 0;

}


//Host functions
float* image_pad(float *im, int m, int n, int patchsize_x, int patchsize_y){

	int size_y = m + (patchsize_y-1);
	int offsety = (patchsize_y-1)/2;
	int size_x = n + (patchsize_x-1);
	int offsetx = (patchsize_x-1)/2;

	//Padding the data for image's borders
	for (int i=0; i<offsety; i++){
		for (int j=offsetx ; j<size_x - offsetx; j++)
			im[i*size_x+j] = im[(2*offsety-1-i)*size_x+j];
	}

	for (int i=0; i<offsety; i++){
		for (int j=offsetx ; j<size_x - offsetx; j++)
			im[(size_y-offsety+i)*size_x+j] = im[(size_y-offsety-i-1)*size_x+j];
	}

	for (int i=0; i<size_y; i++){
		for (int j=0 ; j<offsetx; j++)
			im[i*size_x+j] = im[i*size_x+2*offsetx-1-j];
	}

	for (int i=0; i<size_y; i++){
		for (int j=0 ; j<offsetx; j++)
			im[i*size_x+size_x-offsetx+j] = im[i*size_x+size_x-offsetx-1-j];
	}

	return im;
}

float* gauss_patch(float *gaussian, int offsetx, int offsety, float patchSigma){

	int u=0;

	for (int kx=-offsetx; kx<offsetx+1; kx++){
		for (int ky = -offsety; ky<offsety+1; ky++){
			gaussian[u] = exp(-(ky*ky+kx*kx)/(2*patchSigma*patchSigma));
			u++;
		}
	}

	return gaussian;
}

